#include <stdio.h>
#include <hip/hip_runtime.h>
#include <unistd.h>
#include <signal.h>
#include <stdint.h>
#include <stdlib.h>
#include <pthread.h>
#include <sys/time.h>

#define NUM_THREADS 4
#define MEMPER 0.9
#define SIZE 1024ul
#define REQ_UNDEFINED   '-'
#define REQ_IDLE        ';'
#define REQ_START       'A'

#define RSP_FINISH      'Z'
#define RSP_UNDEFINED   '-'

struct thread_arg{
	int id;
	int size;
	double* doubles;
	double* compare;
};

void *Increment(void *args)
{
	struct thread_arg* arg;	
	int tid;
	int size;
	double* d;
	int work;

	arg = (struct thread_arg*) args;
   	tid = arg->id;
	size = arg->size;
	d = arg->doubles;
	work = size / (8 * 8);
	//printf("Tid: %d\tSize: %d\td:%p\n",tid,size,d);	
	for(int i = 0; i < work; i++){
		d[(tid * work) + i] *= 2;
	}
   	pthread_exit(NULL);
}

__global__ void 
intensive_kernel(unsigned int *cmd){
	int threadId = threadIdx.x + (blockDim.x * blockIdx.x);
	int done;
	double* input;
	double* compare;
	
	while (cmd[8]) {

                if (threadIdx.x == 0 && blockIdx.x == 0) {
			done = cmd[8];
                        if (cmd[0] == REQ_START && cmd[1] != RSP_FINISH) {
                                // we've got a request for a new job
                                // initialize
                                cmd[7] = 1;
                                __threadfence();
                        }
                        else {
                                cmd[7] = 0;
                                cmd[1] = RSP_UNDEFINED;
                                __threadfence();
                        }
                }

                __syncthreads();

                if (cmd[7] == 1) {
                        while(cmd[2] == 0);
                   	
			input = (double*) ((long long)cmd[6]<<32 | cmd[5]);
			compare = (double*) ((long long)cmd[4]<<32 | cmd[3]);     
			if(input[threadId] - compare[threadId] > 0.01){
				input[threadId] = (compare[threadId] * 2) + exp(input[threadId]);
			}	

			if (threadIdx.x == 0 && blockIdx.x == 0) {
				// finitto
                                cmd[0] = REQ_IDLE;
                                cmd[1] = RSP_FINISH;
                                cmd[7] = 0;
                                __threadfence();

                                // host will set #threads equal to 0 after obtaining the results                        
                                while (cmd[2] != 0);
                        }
                }
                __syncthreads();
        }
}


size_t
available_memory(){
	size_t mem_total = 0;
	size_t mem_free = 0;

	hipMemGetInfo(&mem_free, &mem_total);
	printf("Total memory %dMB\tFree Memory %dMB\n",mem_total/(1024*1024),mem_free/(1024*1024));

	return mem_free;	

}


int 
main(int argc, char **argv){
	
	size_t available_mem = 0;
	double *doubles_host;
	double *doubles_device;	
	double *compare_host;
	double *compare_device;
	unsigned int* cmd_h;
	unsigned int* cmd_d;
	int threads;
	int blocks;
	int timeToRun;
	int result;
	hipStream_t stream1;
	hipStream_t stream2;
	struct timeval t1;
	struct timeval t2;
	pthread_t thread[8];
	struct thread_arg args;

	if(argc < 2){
		printf("Usage: stresstest <duration>\n\tduration\tTime stress will run in seconds\n");
		exit(EXIT_FAILURE);
	}

	timeToRun = atoi(argv[1]);

	hipSetDevice(0);
	hipStreamCreate(&stream1);
	hipStreamCreate(&stream2);
	
	available_mem = available_memory() * 0.9;

	printf("Allocating 90%% of the available memory: (%dMB)\n", available_mem/(1024 * 1024));

	hipMalloc((void**)&doubles_device, available_mem/4 * sizeof(char));
	hipMalloc((void**)&compare_device, available_mem/4 * sizeof(char));
	hipMalloc((void**)&cmd_d, 10 * sizeof(unsigned int));
	
	hipHostMalloc((void **)&cmd_h, 10 * sizeof(unsigned int), hipHostMallocDefault);
	hipHostMalloc((void**)&doubles_host, available_mem/4 * sizeof(char), hipHostMallocDefault);
	hipHostMalloc((void**)&compare_host, available_mem/4 * sizeof(char), hipHostMallocDefault);

	srand(time(NULL));
	printf("Initializing buffers...\n");
	for(int i=0; i < available_mem/32; i++){
		doubles_host[i] = i * rand() * 1.8643;
		compare_host[i] = i * rand() * 1.4903;
	}
	printf("Finished initialization of buffers!\n\n");

	cmd_h[0] = REQ_UNDEFINED;
        cmd_h[1] = RSP_UNDEFINED;
        cmd_h[9] = 0;

	hipMemcpy(doubles_device, doubles_host, available_mem/4 * sizeof(char), hipMemcpyHostToDevice);
	hipMemcpy(compare_device, compare_host, available_mem/4 * sizeof(char), hipMemcpyHostToDevice);

	hipMemcpy(cmd_h+3,&(compare_device), sizeof(double*),hipMemcpyHostToHost);
	hipMemcpy(cmd_h+5,&(doubles_device), sizeof(double*),hipMemcpyHostToHost);
	hipMemcpy(cmd_d, cmd_h, 10 * sizeof(unsigned int), hipMemcpyHostToDevice);

	threads = 1024;
	blocks = available_mem/(16 * threads);	

	gettimeofday(&t1, 0);
	printf("Start stressing...\n");
	intensive_kernel<<<blocks,threads,0,stream1>>>(cmd_d);	

	pid_t pid = fork();

	if(pid == 0){
		//child
		
		if(execv("./temperature", argv) == -1){
			printf("Execv failed!\n");
			exit(EXIT_FAILURE);
		} 		
	}
	else if(pid > 0){
		//parent
		
	
		gettimeofday(&t2, 0);	
		while(t2.tv_sec - t1.tv_sec < timeToRun){

	                usleep(10);
	                cmd_h[0] = REQ_START;
	                cmd_h[1] = RSP_UNDEFINED;
	                cmd_h[2] = random() % 512;
			for(int i=0; i < 8; i++ ){
                                args.id = i;
                                args.size = available_mem/4;
                                args.doubles = compare_host;
                         	result = pthread_create(&thread[i], NULL,
                                               Increment, (void *)&args);
                              	if (result){
                 	        	printf("Unable to create thread\n");
                        		exit(-1);
         	               	}
                        }
                        for(int i = 0; i < 8; i++){
	                        pthread_join(thread[i], NULL);
                        }

	                hipMemcpyAsync(doubles_device, doubles_host, available_mem/4 * sizeof(char), hipMemcpyHostToDevice, stream2);
			hipMemcpyAsync(compare_device, compare_host, available_mem/4 * sizeof(char), hipMemcpyHostToDevice, stream2);

	                // first set #threads
	                hipMemcpyAsync(cmd_d+2, cmd_h+2, 1 * sizeof(unsigned int), hipMemcpyHostToDevice, stream2);
	                hipStreamSynchronize(stream2);
	                
	                // set RSP
	                hipMemcpyAsync(cmd_d+1, cmd_h+1, 1 * sizeof(unsigned int), hipMemcpyHostToDevice, stream2);
	                hipStreamSynchronize(stream2);
	
	                // set REQ 
	                hipMemcpyAsync(cmd_d+0, cmd_h+0, 1 * sizeof(unsigned int), hipMemcpyHostToDevice, stream2);
	                hipStreamSynchronize(stream2);
	                
			int ready = 0;
	               	while (((cmd_h[0] == REQ_START) && cmd_h[1] != RSP_FINISH)) {
	                	ready = 1;

	                        // get RSP
	                	hipMemcpyAsync(&cmd_h[1], &cmd_d[1], 1 * sizeof(unsigned int), hipMemcpyDeviceToHost, stream2);
	                	hipStreamSynchronize(stream2);
	                }
	                if (ready == 1) {
	                        // get data
	                        hipMemcpyAsync(doubles_host,doubles_device, available_mem/4 * sizeof(char), hipMemcpyDeviceToHost, stream2);
	                        hipStreamSynchronize(stream2);
                            	printf("Size: %d\tPointer:%p\n",available_mem/4,doubles_host);	
				for(int i=0; i < 8; i++ ){
     					args.id = i;
					args.size = available_mem/4;
					args.doubles = doubles_host;		
      					result = pthread_create(&thread[i], NULL, 
                          			Increment, (void *)&args);
      					if (result){
         					printf("Unable to create thread\n");
         					exit(-1);
      					}
   				}
				for(int i = 0; i < 8; i++){
					//pthread_join(thread[i], NULL);
				}	
				//pthread_exit(NULL);						
				cmd_h[0] = REQ_UNDEFINED;
                                hipMemcpyAsync(&cmd_d[0], &cmd_h[0], 1 * sizeof(unsigned int), hipMemcpyHostToDevice, stream2);
                                hipStreamSynchronize(stream2); 
	                        // notify GPU by setting #threads equal to 0
	                        cmd_h[2] = 0;
	                        hipMemcpyAsync(&cmd_d[2], &cmd_h[2], 1 * sizeof(unsigned int), hipMemcpyHostToDevice, stream2);
	                	hipStreamSynchronize(stream2);
	                }
			gettimeofday(&t2, 0);
	                        //cudaStreamSynchronize(stream2);
	        }
		cmd_h[8] = 1;
		hipMemcpyAsync(&cmd_d[8], &cmd_h[8], 1 * sizeof(unsigned int), hipMemcpyHostToDevice, stream2);
        	hipStreamSynchronize(stream2);

	}
	else{
		//error
		printf("fork() failed!\n");
		exit(EXIT_FAILURE);
	}                                  	

	printf("Finished!\n");
}
